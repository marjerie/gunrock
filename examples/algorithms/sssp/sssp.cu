#include <gunrock/algorithms/sssp.hxx>
#include "sssp_cpu.hxx"  // Reference implementation
#include <gunrock/util/performance.hxx>
#include <gunrock/io/parameters.hxx>

using namespace gunrock;
using namespace memory;

void test_sssp(int num_arguments, char** argument_array) {
  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // IO

  gunrock::io::cli::parameters_t params(num_arguments, argument_array,
                                        "Single Source Shortest Path");

  csr_t csr;
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;

  if (params.binary) {
    csr.read_binary(params.filename);
  } else {
    csr.from_coo(mm.load(params.filename));
  }

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );  // supports row_indices and column_offsets (default = nullptr)

  // --
  // Params and memory allocation

  srand(time(NULL));

  vertex_t n_vertices = G.get_number_of_vertices();

  thrust::device_vector<weight_t> distances(n_vertices);
  thrust::device_vector<vertex_t> predecessors(n_vertices);
  thrust::device_vector<int> edges_visited(1);
  thrust::device_vector<int> vertices_visited(1);
  int search_depth = 0;

  size_t free_byte;
  size_t total_byte;
  hipMemGetInfo(&free_byte, &total_byte);
  double free_db = (double) free_byte;
  double total_db = (double) total_byte;
  double used_db = total_db - free_db ;
  printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
      used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

  std::size_t initial_size = (G.get_number_of_edges() > G.get_number_of_vertices())
              ? G.get_number_of_edges()
              : G.get_number_of_vertices();

  size_t b_to_fill = free_db - initial_size * sizeof(int) * 1.45;
  size_t num_of_elements_to_fill = b_to_fill / sizeof(int);
  std::cout << "num_of_elements_to_fill: " << num_of_elements_to_fill << std::endl;
  thrust::device_vector<int> fill_memory(num_of_elements_to_fill, 0);

  hipMemGetInfo(&free_byte, &total_byte);
  free_db = (double) free_byte;
  total_db = (double) total_byte;
  used_db = total_db - free_db ;
  printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",
      used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

  // Fill memory

  // unsigned long fill_memory_val = 1000000000;

  // size_t free_byte ;
  // size_t total_byte ;
  // hipMemGetInfo( &free_byte, &total_byte );
  // double free_db = (double)free_byte ;
  // double total_db = (double)total_byte ;
  // double used_db = total_db - free_db ;
  // printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n"
  //     ,used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

  // thrust::device_vector<vertex_t> fill_memory(5*fill_memory_val);

  // hipMemGetInfo( &free_byte, &total_byte );
  // free_db = (double)free_byte ;
  // total_db = (double)total_byte ;
  // used_db = total_db - free_db ;
  // printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n"
  //     ,used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

  // Parse sources
  std::vector<int> source_vect;
  gunrock::io::cli::parse_source_string(params.source_string, &source_vect,
                                        n_vertices, params.num_runs);
  // Parse tags
  std::vector<std::string> tag_vect;
  gunrock::io::cli::parse_tag_string(params.tag_string, &tag_vect);

  // --
  // GPU Run

  /// An example of how one can use std::shared_ptr to allocate memory on the
  /// GPU, using a custom deleter that automatically handles deletion of the
  /// memory.
  // std::shared_ptr<weight_t> distances(
  //     allocate<weight_t>(n_vertices * sizeof(weight_t)),
  //     deleter_t<weight_t>());
  // std::shared_ptr<vertex_t> predecessors(
  //     allocate<vertex_t>(n_vertices * sizeof(vertex_t)),
  //     deleter_t<vertex_t>());

  std::vector<float> run_times;
  for (int i = 0; i < source_vect.size(); i++) {
    // Record run times without collecting metrics (due to overhead)
    run_times.push_back(gunrock::sssp::run(
        G, source_vect[i], false, distances.data().get(),
        predecessors.data().get(), edges_visited.data().get(),
        vertices_visited.data().get(), &search_depth));
  }

  // Use memory allocated to fill
  for (int i = fill_memory.size()-20; i < fill_memory.size(); i++)
    fill_memory[i] = 1;

  print::head(distances, 40, "GPU distances");
  std::cout << "GPU Elapsed Time : " << run_times[params.num_runs - 1]
            << " (ms)" << std::endl;

  // --
  // CPU Run

  if (params.validate) {
    thrust::host_vector<weight_t> h_distances(n_vertices);
    thrust::host_vector<vertex_t> h_predecessors(n_vertices);

    float cpu_elapsed = sssp_cpu::run<csr_t, vertex_t, edge_t, weight_t>(
        csr, source_vect.back(), h_distances.data(), h_predecessors.data());

    int n_errors =
        util::compare(distances.data().get(), h_distances.data(), n_vertices);

    print::head(h_distances, 40, "CPU Distances");

    std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
    std::cout << "Number of errors : " << n_errors << std::endl;
  }

  // --
  // Run performance evaluation

  if (params.collect_metrics) {
    std::vector<int> edges_visited_vect;
    std::vector<int> search_depth_vect;
    std::vector<int> nodes_visited_vect;

    vertex_t n_edges = G.get_number_of_edges();

    for (int i = 0; i < source_vect.size(); i++) {
      float metrics_run_time = gunrock::sssp::run(
          G, source_vect[i], params.collect_metrics, distances.data().get(),
          predecessors.data().get(), edges_visited.data().get(),
          vertices_visited.data().get(), &search_depth);

      thrust::host_vector<int> h_edges_visited = edges_visited;
      thrust::host_vector<int> h_vertices_visited = vertices_visited;

      edges_visited_vect.push_back(h_edges_visited[0]);
      nodes_visited_vect.push_back(h_vertices_visited[0]);
      search_depth_vect.push_back(search_depth);
    }

    gunrock::util::stats::get_performance_stats(
        edges_visited_vect, nodes_visited_vect, n_edges, n_vertices,
        search_depth_vect, run_times, "sssp", params.filename, "market",
        params.json_dir, params.json_file, source_vect, tag_vect, num_arguments,
        argument_array);
  }
}

int main(int argc, char** argv) {
  test_sssp(argc, argv);
}
